#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <cstdlib>
#include "math.h"
#include "CudaSamples.h"
#define NWarps 32

__global__ void HelloWorld()
{
	//Global index build with blocks and threads
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        int tId = threadIdx.x;
        int BlockId=blockIdx.x;
	printf("Hi from thread number %d, Block number %d, global index %d\n",tId,BlockId,index);	
}
__global__ void HelloWorld2D()
{
	//2D global index
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        int tId_x = threadIdx.x;
        int BlockId_x=blockIdx.x;
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        int tId_y = threadIdx.y;
        int BlockId_y=blockIdx.y;

	printf("Hi from thread X number %d, Block X number %d, global index X number%d\n",tId_x,BlockId_x,index_x);
	printf("Hi from thread Y number %d, Block Y number %d, global index Y number%d\n\n",tId_y,BlockId_y,index_y);	
}

void CallHelloWorld(int N)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(N)),1,1);
//1d blocks per grid
	const dim3 grid(iDivUp(N,NWarps),1,1);
//Kernel launch
        HelloWorld<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

void CallHelloWorld2D(int Nx,int Ny)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Nx)),min(float(NWarps),float(Ny)),1);
//2d blocks per grid
	const dim3 grid(iDivUp(Nx,NWarps),iDivUp(Ny,NWarps),1);
//Kernel launch
        HelloWorld2D<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

__global__ void KernelSumUp(int *Input1, int *Input2, int *Output, int Dim)
{
	//Global index build with blocks and threads
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        if(index<Dim)
         {
          Output[index]=Input1[index]+Input2[index];
         }	
}
__global__ void KernelSumUp2D(int *Input1, int *Input2, int *Output, int Width, int Height)
{
	//2D global Index mapping 
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        //
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        //
        if(index_x<Width && index_y<Height)
         {
        //2d mapping
        int Index_xy=index_y*Width+index_x; 
        Output[Index_xy]=Input1[Index_xy]+Input2[Index_xy];		
         }
}

void SumUp(int *Input1, int *Input2, int *Output, int Dim)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Dim)),1,1);
//blocks per grid
	const dim3 grid(iDivUp(Dim,NWarps),1,1);
//Kernel launch
        KernelSumUp<<<grid,block>>>(Input1, Input2, Output, Dim);
//Synchronize threads
        hipDeviceSynchronize();
}

void SumUp2D(int *Input1, int *Input2, int *Output, int Width, int Height)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Width)),min(float(NWarps),float(Height)),1);
//2d blocks per grid
	const dim3 grid(iDivUp(Width,NWarps),iDivUp(Height,NWarps),1);
//Kernel launch
        KernelSumUp2D<<<grid,block>>>(Input1, Input2, Output, Width, Height);
//Synchronize threads
        hipDeviceSynchronize();
}

