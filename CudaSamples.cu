#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <cstdlib>
#include "math.h"
#include "CudaSamples.h"
#define NWarps 32

// Inputs array with random numbers [1,9999].
void Randomize(int *array, int N){
  srand (time(NULL)); // initialization.

  for(int i = 0; i < N; i++){
    array[i] = (int) rand() % 10000;
  }
}
//
//
int CompareOutputs(int *Input1,int *Input2, int *Output, int N) {
//
       int Diff=0;
	for(int i = 0; i < N ; i++) {
		Diff += ((Input1[i] + Input2[i] - Output[i])*(Input1[i] + Input2[i] - Output[i]));
	}
        return Diff;	
}
//
__global__ void HelloWorld()
{
	//Global index build with blocks and threads
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        int tId = threadIdx.x;
        int BlockId=blockIdx.x;
	printf("Hi from thread number %d, Block number %d, global index %d\n",tId,BlockId,index);	
}
__global__ void HelloWorld2D()
{
	//2D global index
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        int tId_x = threadIdx.x;
        int BlockId_x=blockIdx.x;
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        int tId_y = threadIdx.y;
        int BlockId_y=blockIdx.y;

	printf("Hi from thread X number %d, Block X number %d, global index X number%d\n",tId_x,BlockId_x,index_x);
	printf("Hi from thread Y number %d, Block Y number %d, global index Y number%d\n\n",tId_y,BlockId_y,index_y);	
}

void CallHelloWorld(int N)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(N)),1,1);
//1d blocks per grid
	const dim3 grid(iDivUp(N,NWarps),1,1);
//Kernel launch
        HelloWorld<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

void CallHelloWorld2D(int Nx,int Ny)
{
//N Threads per block 
	const dim3 block(min(NWarps,Nx),min(NWarps,Ny),1);
//2d blocks per grid
	const dim3 grid(iDivUp(Nx,NWarps),iDivUp(Ny,NWarps),1);
//Kernel launch
        HelloWorld2D<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

__global__ void KernelSumUp(int *Input1, int *Input2, int *Output, int Dim)
{
	//Global index build with blocks and threads
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        if(index<Dim)
         {
          Output[index]=Input1[index]+Input2[index];
          //printf("Index %d, Input1[index] %d Input2[index] %d\n",index,Input1[index],Input2[index]);
         }	
}
__global__ void KernelSumUp2D(int *Input1, int *Input2, int *Output, int Width, int Height)
{
	//2D global Index mapping 
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        //
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        //
        if(index_x<Width && index_y<Height)
         {
        //2d mapping
        int Index_xy=index_y*Width+index_x; 
        Output[Index_xy]=Input1[Index_xy]+Input2[Index_xy];		
         }
}

int SumUp(int Dim)
{
   //
   int NBytes = sizeof(int)*Dim;
   //
   int *Input1Dev, *Input2Dev, *OutputDev;
   //
   int *Input1, *Input2, *Output;
   //
   Input1 = new int[Dim];
   Input2 = new int[Dim];
   Output = new int[Dim];

   Randomize(Input1, Dim);

   Randomize(Input2, Dim);


   (hipMalloc(&Input1Dev,NBytes));
   (hipMalloc(&Input2Dev,NBytes));
   (hipMalloc(&OutputDev,NBytes));
   //
   (hipMemcpy(Input1Dev,Input1,NBytes,hipMemcpyHostToDevice));
   (hipMemcpy(Input2Dev,Input2,NBytes,hipMemcpyHostToDevice));   
   //
   
//N Threads per block 
	const dim3 block(min(NWarps,Dim),1,1);
//blocks per grid
	const dim3 grid(iDivUp(Dim,NWarps),1,1);
//Kernel launch
        KernelSumUp<<<grid,block>>>(Input1Dev, Input2Dev, OutputDev, Dim);
//Synchronize threads
         hipDeviceSynchronize();
//        
 (hipMemcpy(Output,OutputDev,NBytes,hipMemcpyDeviceToHost));   
//
         hipFree(Input1Dev);
         hipFree(Input2Dev);
         hipFree(OutputDev);
//
int Check=CompareOutputs(Input1,Input2, Output,Dim); 

         delete [] Input1;
         delete [] Input2;
         delete [] Output;
         
         return Check;
}

int SumUp2D(int Width, int Height)
{
   int NBytes = sizeof(int)*Width*Height;
//
   int *Input1, *Input2, *Output;
//
   int Dim=Width*Height;
   Input1 = new int[Dim];
   Input2 = new int[Dim];
   Output = new int[Dim];
//
   int *Input1Dev, *Input2Dev, *OutputDev;
//
   (hipMalloc(&Input1Dev,NBytes));
   (hipMalloc(&Input2Dev,NBytes));
   (hipMalloc(&OutputDev,NBytes));
//
   Randomize(Input1, Dim);
//
   Randomize(Input2, Dim);

   (hipMemcpy(Input1Dev,Input1,NBytes,hipMemcpyHostToDevice));
   (hipMemcpy(Input2Dev,Input2,NBytes,hipMemcpyHostToDevice));  
//N Threads per block 
	const dim3 block(min(NWarps,Width),min(NWarps,Height),1);
//2d blocks per grid
	const dim3 grid(iDivUp(Width,NWarps),iDivUp(Height,NWarps),1);
//Kernel launch
        KernelSumUp2D<<<grid,block>>>(Input1Dev, Input2Dev, OutputDev, Width, Height);
//Synchronize threads
        hipDeviceSynchronize();
//
    (hipMemcpy(Output,OutputDev,NBytes,hipMemcpyDeviceToHost)); 

int Check=CompareOutputs(Input1,Input2, Output,Dim); 

         delete [] Input1;
         delete [] Input2;
         delete [] Output;
         
         return Check;        
}

