#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <cstdlib>
#include "math.h"
#include "CudaSamples.h"
#define NWarps 32

// Inputs array with random numbers [1,9999].
void Randomize(int *array, int N){
  srand (time(NULL)); // initialization.

  for(int i = 0; i < N; i++){
    array[i] = (int) rand() % 10000;
  }
}
//
//
int CompareOutputs(int *Input1,int *Input2, int *Output, int N) {
//
       int Diff=0;
	for(int i = 0; i < N ; i++) {
		Diff += (Input1[i] + Input2[i] - Output[i])*(Input1[i] + Input2[i] - Output[i]);
	}
      if(Diff==0){return 1;}
      else{return 0;}	
}
//
__global__ void HelloWorld()
{
	//Global index build with blocks and threads
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        int tId = threadIdx.x;
        int BlockId=blockIdx.x;
	printf("Hi from thread number %d, Block number %d, global index %d\n",tId,BlockId,index);	
}
__global__ void HelloWorld2D()
{
	//2D global index
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        int tId_x = threadIdx.x;
        int BlockId_x=blockIdx.x;
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        int tId_y = threadIdx.y;
        int BlockId_y=blockIdx.y;

	printf("Hi from thread X number %d, Block X number %d, global index X number%d\n",tId_x,BlockId_x,index_x);
	printf("Hi from thread Y number %d, Block Y number %d, global index Y number%d\n\n",tId_y,BlockId_y,index_y);	
}

void CallHelloWorld(int N)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(N)),1,1);
//1d blocks per grid
	const dim3 grid(iDivUp(N,NWarps),1,1);
//Kernel launch
        HelloWorld<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

void CallHelloWorld2D(int Nx,int Ny)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Nx)),min(float(NWarps),float(Ny)),1);
//2d blocks per grid
	const dim3 grid(iDivUp(Nx,NWarps),iDivUp(Ny,NWarps),1);
//Kernel launch
        HelloWorld2D<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

__global__ void KernelSumUp(int *Input1, int *Input2, int *Output, int Dim)
{
	//Global index build with blocks and threads
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        if(index<Dim)
         {
          Output[index]=Input1[index]+Input2[index];
         }	
}
__global__ void KernelSumUp2D(int *Input1, int *Input2, int *Output, int Width, int Height)
{
	//2D global Index mapping 
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        //
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        //
        if(index_x<Width && index_y<Height)
         {
        //2d mapping
        int Index_xy=index_y*Width+index_x; 
        Output[Index_xy]=Input1[Index_xy]+Input2[Index_xy];		
         }
}

int SumUp(int Dim)
{
   //
   int NBytes = sizeof(int)*Dim;
   //
   int *Input1Dev, *Input2Dev, *OutputDev;
   //
   int *Input1, *Input2, *Output;
   //
   Input1 = (int*) malloc (Dim);
   Input2 = (int*) malloc (Dim);
   Output = (int*) malloc (Dim);

   Randomize(Input1, Dim);

   Randomize(Input2, Dim);


   (hipMalloc<int>(&Input1Dev,NBytes));
   (hipMalloc<int>(&Input2Dev,NBytes));
   (hipMalloc<int>(&OutputDev,NBytes));
   //
   (hipMemcpy(Input1Dev,Input1,NBytes,hipMemcpyHostToDevice));
   (hipMemcpy(Input2Dev,Input2,NBytes,hipMemcpyHostToDevice));   
   //
   
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Dim)),1,1);
//blocks per grid
	const dim3 grid(iDivUp(Dim,NWarps),1,1);
//Kernel launch
        KernelSumUp<<<grid,block>>>(Input1Dev, Input2Dev, OutputDev, Dim);
//Synchronize threads
         hipDeviceSynchronize();
//        
 (hipMemcpy(Output,OutputDev,NBytes,hipMemcpyDeviceToHost));   
//
         hipFree(Input1Dev);
         hipFree(Input2Dev);
         hipFree(OutputDev);
//
         free(Input1);
         free(Input2);
         free(Output);
return CompareOutputs(Input1,Input2, Output,Dim); 
}

int SumUp2D(int Width, int Height)
{
   int NBytes = sizeof(int)*Width*Height;
//
   int *Input1, *Input2, *Output;
//
   Input1 = (int*) malloc (Width*Height);
   Input2 = (int*) malloc (Width*Height);
   Output = (int*) malloc (Width*Height);
//
   int *Input1Dev, *Input2Dev, *OutputDev;
//
   (hipMalloc<int>(&Input1Dev,NBytes));
   (hipMalloc<int>(&Input2Dev,NBytes));
   (hipMalloc<int>(&OutputDev,NBytes));
//
   Randomize(Input1, Width*Height);
//
   Randomize(Input2, Width*Height);

   (hipMemcpy(Input1Dev,Input1,NBytes,hipMemcpyHostToDevice));
   (hipMemcpy(Input2Dev,Input2,NBytes,hipMemcpyHostToDevice));  
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Width)),min(float(NWarps),float(Height)),1);
//2d blocks per grid
	const dim3 grid(iDivUp(Width,NWarps),iDivUp(Height,NWarps),1);
//Kernel launch
        KernelSumUp2D<<<grid,block>>>(Input1Dev, Input2Dev, OutputDev, Width, Height);
//Synchronize threads
        hipDeviceSynchronize();
//
    (hipMemcpy(Output,OutputDev,NBytes,hipMemcpyDeviceToHost)); 


         hipFree(Input1Dev);
         hipFree(Input2Dev);
         hipFree(OutputDev);   
//
         free(Input1);
         free(Input2);
         free(Output);    
// 
return CompareOutputs(Input1,Input2, Output,Width*Height);         
}

