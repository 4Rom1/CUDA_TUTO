#include "hip/hip_runtime.h"
#include "CudaSamples.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

// Inputs array with random numbers [1,9999].
void Randomize(int *array, int N) {
  srand(time(NULL)); // initialization.

  for (int i = 0; i < N; i++) {
    array[i] = (int)rand() % 10000;
  }
}
//
//
int CompareOutputs(int *Input1, int *Input2, int *Output, int N) {
  //
  int Diff = 0;
  for (int i = 0; i < N; i++) {
    Diff += ((Input1[i] + Input2[i] - Output[i]) *
             (Input1[i] + Input2[i] - Output[i]));
  }
  return Diff;
}
//
__global__ void HelloWorld() {
  // Global index build with blocks and threads
  const int index = threadIdx.x + blockDim.x * blockIdx.x;
  const int tId = threadIdx.x;
  const int BlockId = blockIdx.x;
  printf("Hi from thread number %d, Block number %d, global index %d\n", tId,
         BlockId, index);
}
__global__ void HelloWorld2D() {
  // 2D global index
  const int index_x = threadIdx.x + blockDim.x * blockIdx.x;
  const int tId_x = threadIdx.x;
  const int BlockId_x = blockIdx.x;
  const int index_y = threadIdx.y + blockDim.y * blockIdx.y;
  const int tId_y = threadIdx.y;
  const int BlockId_y = blockIdx.y;

  printf("Hi from thread X number %d, Block X number %d, global index X "
         "number%d\n",
         tId_x, BlockId_x, index_x);
  printf("Hi from thread Y number %d, Block Y number %d, global index Y "
         "number%d\n\n",
         tId_y, BlockId_y, index_y);
}

void CallHelloWorld(int N) {
  // N Threads per block
  const dim3 block(min(float(NWarps), float(N)), 1, 1);
  // 1d blocks per grid
  const dim3 grid(iDivUp(N, NWarps), 1, 1);
  // Kernel launch
  HelloWorld<<<grid, block>>>();
  // Synchronize threads
  GPU_ERROR_CHECK(hipDeviceSynchronize())
}

void CallHelloWorld2D(int Nx, int Ny) {
  // N Threads per block
  const dim3 block(min(NWarps, Nx), min(NWarps, Ny), 1);
  // 2d blocks per grid
  const dim3 grid(iDivUp(Nx, NWarps), iDivUp(Ny, NWarps), 1);
  // Kernel launch
  HelloWorld2D<<<grid, block>>>();
  // Synchronize threads
  GPU_ERROR_CHECK(hipDeviceSynchronize())
}

__global__ void KernelSumUp(int *Input1, int *Input2, int *Output, int Dim) {
  // Global index build with blocks and threads
  const int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index < Dim) {
    Output[index] = Input1[index] + Input2[index];
  }
}
__global__ void KernelSumUp2D(int *Input1, int *Input2, int *Output, int Width,
                              int Height) {
  // 2D global Index mapping
  const int index_x = threadIdx.x + blockDim.x * blockIdx.x;
  //
  const int index_y = threadIdx.y + blockDim.y * blockIdx.y;
  //
  if (index_x < Width && index_y < Height) {
    // 2d mapping
    const int Index_xy = index_y * Width + index_x;
    Output[Index_xy] = Input1[Index_xy] + Input2[Index_xy];
  }
}

int SumUp(int Dim) {
  struct timeval begin, end;
  //
  int NBytes = sizeof(int) * Dim;
  //
  int *Input1Dev, *Input2Dev, *OutputDev;
  //
  int *Input1, *Input2, *Output;
  //
  Input1 = new int[Dim];
  Input2 = new int[Dim];
  Output = new int[Dim];

  Randomize(Input1, Dim);

  Randomize(Input2, Dim);

  gettimeofday(&begin, NULL);

  (hipMalloc(&Input1Dev, NBytes));
  (hipMalloc(&Input2Dev, NBytes));
  (hipMalloc(&OutputDev, NBytes));
  //
  gettimeofday(&end, NULL);
  unsigned int delta_time = TIME_DIFFS(begin, end);
  printf("time spent for allocation %u micros\n", delta_time);
  gettimeofday(&begin, NULL);
  (hipMemcpy(Input1Dev, Input1, NBytes, hipMemcpyHostToDevice));
  (hipMemcpy(Input2Dev, Input2, NBytes, hipMemcpyHostToDevice));
  //
  gettimeofday(&end, NULL);
  delta_time = TIME_DIFFS(begin, end);
  printf("time spent for synchronous copy %u micros\n", delta_time);
  // N Threads per block
  const dim3 block(min(NWarps, Dim), 1, 1);
  // blocks per grid
  const dim3 grid(iDivUp(Dim, NWarps), 1, 1);
  // Kernel launch
  gettimeofday(&begin, NULL);
  KernelSumUp<<<grid, block>>>(Input1Dev, Input2Dev, OutputDev, Dim);
  // Synchronize threads
  GPU_ERROR_CHECK(hipDeviceSynchronize())
  //
  gettimeofday(&end, NULL);
  delta_time = TIME_DIFFS(begin, end);
  printf("time spent for one kernel launch synchronized %u micros\n",
         delta_time);
  GPU_ERROR_CHECK(hipMemcpy(Output, OutputDev, NBytes, hipMemcpyDeviceToHost))
  //
  hipFree(Input1Dev);
  hipFree(Input2Dev);
  hipFree(OutputDev);
  //
  int Check = CompareOutputs(Input1, Input2, Output, Dim);

  delete[] Input1;
  delete[] Input2;
  delete[] Output;

  return Check;
}
int SumUpStreams(int Dim) {
  struct timeval begin, end;
  //
  int NBytes = sizeof(int) * Dim;
  //
  int *Input1Dev, *Input2Dev, *OutputDev;
  //
  int *Input1, *Input2, *Output;
  //
  hipStream_t stream1, stream2;
  //
  gettimeofday(&begin, NULL);
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  gettimeofday(&end, NULL);
  unsigned int delta_time = TIME_DIFFS(begin, end);
  printf("time spent for stream creation %u micros\n", delta_time);

  Input1 = new int[Dim];
  Input2 = new int[Dim];
  Output = new int[Dim];

  Randomize(Input1, Dim);

  Randomize(Input2, Dim);

  gettimeofday(&begin, NULL);
  hipMalloc(&Input1Dev, NBytes);
  hipMalloc(&Input2Dev, NBytes);
  hipMalloc(&OutputDev, NBytes);
  gettimeofday(&end, NULL);
  delta_time = TIME_DIFFS(begin, end);
  printf("time spent for allocation %u micros\n", delta_time);
  gettimeofday(&begin, NULL);
  // Asynchronous copy in parrallel
  hipMemcpyAsync(Input1Dev, Input1, NBytes, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(Input2Dev, Input2, NBytes, hipMemcpyHostToDevice, stream2);
  // Synchronize threads and streams
  GPU_ERROR_CHECK(hipDeviceSynchronize())
  gettimeofday(&end, NULL);
  delta_time = TIME_DIFFS(begin, end);
  printf("time spent for asynchronous copy %u micros\n", delta_time);
  // N/2 Threads per block
  const dim3 block(min(NWarps, iDivUp(Dim, 2)), 1, 1);
  // blocks per grid
  const dim3 grid(iDivUp(iDivUp(Dim, 2), NWarps), 1, 1);
  // Kernel launch both streams in parallel
  gettimeofday(&begin, NULL);
  KernelSumUp<<<grid, block, 0, stream1>>>(Input1Dev, Input2Dev, OutputDev,
                                           Dim / 2);
  KernelSumUp<<<grid, block, 0, stream2>>>(&Input1Dev[Dim / 2],
                                           &Input2Dev[Dim / 2],
                                           &OutputDev[Dim / 2], iDivUp(Dim, 2));
  // Synchronize threads and streams
  GPU_ERROR_CHECK(hipDeviceSynchronize())

  gettimeofday(&end, NULL);
  delta_time = TIME_DIFFS(begin, end);
  printf("time spent for 2 non synchronous kernel launch %u micros\n",
         delta_time);
  //
  GPU_ERROR_CHECK(hipMemcpy(Output, OutputDev, NBytes, hipMemcpyDeviceToHost))
  //
  hipFree(Input1Dev);
  hipFree(Input2Dev);
  hipFree(OutputDev);
  //
  int Check = CompareOutputs(Input1, Input2, Output, Dim);

  delete[] Input1;
  delete[] Input2;
  delete[] Output;

  return Check;
}

int SumUp2D(int Width, int Height) {
  int NBytes = sizeof(int) * Width * Height;
  //
  int *Input1, *Input2, *Output;
  //
  int Dim = Width * Height;
  Input1 = new int[Dim];
  Input2 = new int[Dim];
  Output = new int[Dim];
  //
  int *Input1Dev, *Input2Dev, *OutputDev;
  //
  GPU_ERROR_CHECK(hipMalloc(&Input1Dev, NBytes))
  GPU_ERROR_CHECK(hipMalloc(&Input2Dev, NBytes))
  GPU_ERROR_CHECK(hipMalloc(&OutputDev, NBytes))
  //
  Randomize(Input1, Dim);
  //
  Randomize(Input2, Dim);

  GPU_ERROR_CHECK(hipMemcpy(Input1Dev, Input1, NBytes, hipMemcpyHostToDevice))
  GPU_ERROR_CHECK(hipMemcpy(Input2Dev, Input2, NBytes, hipMemcpyHostToDevice))
  // N Threads per block
  const dim3 block(min(NWarps, Width), min(NWarps, Height), 1);
  // 2d blocks per grid
  const dim3 grid(iDivUp(Width, NWarps), iDivUp(Height, NWarps), 1);
  // Kernel launch
  KernelSumUp2D<<<grid, block>>>(Input1Dev, Input2Dev, OutputDev, Width,
                                 Height);
  // Synchronize threads
  GPU_ERROR_CHECK(hipDeviceSynchronize())
  //
  GPU_ERROR_CHECK(hipMemcpy(Output, OutputDev, NBytes, hipMemcpyDeviceToHost))

  int Check = CompareOutputs(Input1, Input2, Output, Dim);

  delete[] Input1;
  delete[] Input2;
  delete[] Output;

  return Check;
}

__global__ void ParSqrtExp(float *x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int ii = tid; ii < n; ii += blockDim.x * gridDim.x) {
    x[ii] = float(tid);
  }
}

void parallelSqrtExp(float *data, 
  std::vector<hipStream_t> streams, int N, int num_streams) {

  for (int i = 0; i < num_streams; i++) {
    ParSqrtExp<<<1, 1024, 0, streams[i]>>>(&data[i * N], N);
  }
}
