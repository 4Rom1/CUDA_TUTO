#include "CudaSamples.h"
#include <pthread.h>
#include <stdio.h>

int main(int argc, const char *argv[]) {
  int N = 4;
  //
  printf("Usage : MultiStream N\n");
  printf("N : maximal global dimension\n");
  printf("Profile nvprof --export-profile FileName.prof ./MultiStream N\n");
  //
  if (argc >= 2) {
    N = atoi(argv[1]);
  }
  //
  const int num_streams = 8;
  hipStream_t streams[num_streams];
  //
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }
  //
  float *data;
  //

  //
  hipMalloc(&data, N * sizeof(float) * num_streams);
  for (int i = 0; i < num_streams; i++) {
    ParSqrtExp<<<1, 1024, 0, streams[i]>>>(&data[i * N], N);
  }
  //
  hipDeviceSynchronize();
  hipFree(data);
  return 0;
}
