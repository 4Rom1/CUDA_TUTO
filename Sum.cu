#include "hip/hip_runtime.h"
#include <cstdio>

#include <cstdlib>

#include <string> 

#include <iostream>

#include <sys/time.h>

#include <unistd.h>

#include <sstream>

#include "CudaSamples.h"
//
#include <hip/hip_runtime.h>
//
#include <sys/time.h>
//
#define TIME_DIFFS(t1, t2) \
t2.tv_usec - t1.tv_usec  
//
int main (int argc,const char* argv[])
{
int Nx=4,Ny=4;

struct timeval begin, end;
uint32_t delta_time=0;

printf("Usage : ./Sum N\n");
printf("N : maximal global dimension or\n");
printf("./Sum Nx Ny\n");
printf("Nx max global dimension x, Ny max global dimension y\n");
int Check1=0;
int Check2=0;
  if(argc >= 2)
     {
 Nx=atoi(argv[1]);
     }
//Time initialization
  gettimeofday(&begin,NULL);
 printf("Calling sum up 1d, N=%d\n",Nx);
 Check1=SumUp(Nx);
  if(Check1)
  {printf("Sum non identique, Diff square = %d\n",Check1);}
  else
  {printf("Sum identiques\n");}
//Time end
     gettimeofday(&end,NULL);
     delta_time=TIME_DIFFS(begin, end); 
//Define the maximum size of active space B and residual space
    printf("time spent synchronous sum %u micros\n", delta_time);
//Time initialization
  gettimeofday(&begin,NULL);
 printf("Calling sum up 1d asynchronous, N=%d\n",Nx);
 Check1=SumUpStreams(Nx);
//Time end
     gettimeofday(&end,NULL);
     delta_time=TIME_DIFFS(begin, end); 
//Define the maximum size of active space B and residual space
    printf("time spent asynchronous sum %u micros\n", delta_time);
//
  if(Check1)
  {printf("Sum non identique, Diff square = %d\n",Check1);}
  else
  {printf("Sum identiques\n");}

  
  if(argc >= 3)
     { 
   Ny=atoi(argv[2]);
   printf("Calling sum up 2D Nx=%d, Ny=%d\n",Nx,Ny);
   Check2=SumUp2D(Nx,Ny);
  if(Check2)
  {printf("Sum non identique, Diff square = %d\n",Check2);}
  else
  {printf("Sum identiques\n");}
     }   

}
