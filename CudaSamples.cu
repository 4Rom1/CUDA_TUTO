#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <cstdlib>
#include "math.h"
#include "CudaSamples.h"
#define NWarps 32


__global__ void HelloWorld()
{
	//2D Index of current thread
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        int tId = threadIdx.x;
        int BlockId=blockIdx.x;
	printf("Hi from thread number %d, Block number %d, global index %d\n",tId,BlockId,index);	
}
void CallHelloWorld(int N)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(N)),1,1);
//1 block per grid
	const dim3 grid(iDivUp(N,NWarps),1,1);
//Kernel launch
        HelloWorld<<<grid,block>>>();
}
