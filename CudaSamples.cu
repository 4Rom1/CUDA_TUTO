#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <cstdlib>
#include "math.h"
#include "CudaSamples.h"
#define NWarps 32

__global__ void HelloWorld()
{
	//2D Index of current thread
        int index = threadIdx.x + blockDim.x * blockIdx.x;
        int tId = threadIdx.x;
        int BlockId=blockIdx.x;
	printf("Hi from thread number %d, Block number %d, global index %d\n",tId,BlockId,index);	
}
__global__ void HelloWorld2D()
{
	//2D Index of current thread
        int index_x = threadIdx.x + blockDim.x * blockIdx.x;
        int tId_x = threadIdx.x;
        int BlockId_x=blockIdx.x;
        int index_y = threadIdx.y + blockDim.y * blockIdx.y;
        int tId_y = threadIdx.y;
        int BlockId_y=blockIdx.y;

	printf("Hi from thread X number %d, Block X number %d, global index X number%d\n",tId_x,BlockId_x,index_x);
	printf("Hi from thread Y number %d, Block Y number %d, global index Y number%d\n",tId_y,BlockId_y,index_y);	
}

void CallHelloWorld(int N)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(N)),1,1);
//1 block per grid
	const dim3 grid(iDivUp(N,NWarps),1,1);
//Kernel launch
        HelloWorld<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

void CallHelloWorld2D(int Nx,int Ny)
{
//N Threads per block 
	const dim3 block(min(float(NWarps),float(Nx)),min(float(NWarps),float(Ny)),1);
//1 block per grid
	const dim3 grid(iDivUp(Nx,NWarps),iDivUp(Ny,NWarps),1);
//Kernel launch
        HelloWorld2D<<<grid,block>>>();
//Synchronize threads
        hipDeviceSynchronize();
}

