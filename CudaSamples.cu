#include <hip/hip_runtime.h>

#include "stdio.h"
#include <cstdlib>


__global__ void HelloWorld()
{
	//2D Index of current thread
	const int ThreadXIndex = threadIdx.x;
	printf("Hi from thread number %d\n",ThreadXIndex);	
}
void CallHelloWorld(int N)
{
//N Threads per block 
	const dim3 block(N,1);
//1 block per grid
	const dim3 grid(1);
//Kernel launch
        HelloWorld<<<grid,block>>>();
}
